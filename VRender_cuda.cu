#include "hip/hip_runtime.h"
#include <algorithm>
#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "VRender_cuda_kernel.cuh"
#include "Cloud.h"


int iDivUp( int a, int b ){ return (a % b != 0) ? (a / b + 1) : (a / b); }


// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


extern "C"
void allocateMemory( Cloud *cloud, int device, hipExtent volumeSize, uint imageW, uint imageH )
{
    hipSetDevice(device);

    float cudatime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    checkCudaErrors( hipMalloc( (void**) &cellStart, cloud->world.count * sizeof(uint) ) );
    checkCudaErrors( hipMalloc( (void**) &cellEnd, cloud->world.count * sizeof(uint) ) );
    checkCudaErrors( hipMemset( cellStart, 0xffffffff, cloud->world.count * sizeof(uint) ) );
    checkCudaErrors( hipMemset( cellEnd, 0xffffffff, cloud->world.count * sizeof(uint) ) );

    checkCudaErrors( hipMalloc( (void**) &d_red, cloud->world.count ) );
    checkCudaErrors( hipMalloc( (void**) &d_green, cloud->world.count ) );
    checkCudaErrors( hipMalloc( (void**) &d_blue, cloud->world.count ) );
    checkCudaErrors( hipMemset( d_red, 0, cloud->world.count ) );
    checkCudaErrors( hipMemset( d_green, 0, cloud->world.count ) );
    checkCudaErrors( hipMemset( d_blue, 0, cloud->world.count ) );

    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( d_pcl), &cloud->pcl, sizeof(PCListData) ) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( d_world), &cloud->world, sizeof(WORLD) ) );

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();

    // RED
    checkCudaErrors( hipMalloc3DArray( &d_redArray, &channelDesc, volumeSize ) );
    redParams.srcPtr   =   make_hipPitchedPtr( d_red, volumeSize.width, volumeSize.width, volumeSize.height );
    redParams.dstArray =   d_redArray;
    redParams.extent   =   volumeSize;
    redParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &redParams ) );

    texRed.normalized = true;
    texRed.filterMode = hipFilterModeLinear;
    texRed.addressMode[0] = hipAddressModeClamp;
    texRed.addressMode[1] = hipAddressModeClamp;
    texRed.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texRed, d_redArray, channelDesc ) );

    // GREEN
    checkCudaErrors( hipMalloc3DArray( &d_greenArray, &channelDesc, volumeSize ) );
    greenParams.srcPtr   =   make_hipPitchedPtr( d_green, volumeSize.width, volumeSize.width, volumeSize.height );
    greenParams.dstArray =   d_greenArray;
    greenParams.extent   =   volumeSize;
    greenParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &greenParams ) );

    texGreen.normalized = true;
    texGreen.filterMode = hipFilterModeLinear;
    texGreen.addressMode[0] = hipAddressModeClamp;
    texGreen.addressMode[1] = hipAddressModeClamp;
    texGreen.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texGreen, d_greenArray, channelDesc ) );

    // BLUE
    checkCudaErrors( hipMalloc3DArray( &d_blueArray, &channelDesc, volumeSize ) );
    blueParams.srcPtr   =   make_hipPitchedPtr( d_blue, volumeSize.width, volumeSize.width, volumeSize.height );
    blueParams.dstArray =   d_blueArray;
    blueParams.extent   =   volumeSize;
    blueParams.kind     =   hipMemcpyDeviceToDevice;
    checkCudaErrors( hipMemcpy3D( &blueParams ) );

    texBlue.normalized = true;
    texBlue.filterMode = hipFilterModeLinear;
    texBlue.addressMode[0] = hipAddressModeClamp;
    texBlue.addressMode[1] = hipAddressModeClamp;
    texBlue.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors( hipBindTextureToArray( texBlue, d_blueArray, channelDesc ) );

    // OUTPUT BUFFER
    checkCudaErrors( hipMalloc( (void**) &d_volume, imageW * imageH * 3 ) );
    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 ) );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &cudatime, start, stop );
    printf("\n ||| TIME - GPU Memory Allocation: %f ms\n", cudatime);
}



extern "C"
void updateVRenderColorMaps( Cloud * cloud )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( d_pcl), &cloud->pcl, sizeof(PCListData) ) );

    h_pos = new float3[cloud->pcl.count];
    h_color = new uint3[cloud->pcl.count];
    checkCudaErrors( hipMalloc( (void**) &d_pos, cloud->pcl.count * sizeof(float3) ) );
    checkCudaErrors( hipMalloc( (void**) &d_color, cloud->pcl.count * sizeof(uint3) ) );
    std::copy( cloud->position.begin(), cloud->position.end(), h_pos );
    checkCudaErrors( hipMemcpy( d_pos, h_pos, cloud->pcl.count * sizeof(float3), hipMemcpyHostToDevice ) );
    std::copy( cloud->rgb.begin(), cloud->rgb.end(), h_color );
    checkCudaErrors( hipMemcpy( d_color, h_color, cloud->pcl.count * sizeof(uint3), hipMemcpyHostToDevice ) );

    uint numThreads, numBlocks;
    computeGridSize( cloud->pcl.count, 256, numBlocks, numThreads);

    checkCudaErrors( hipMalloc( (void**) &gridHash, cloud->pcl.count * sizeof(uint) ) );
    checkCudaErrors( hipMalloc( (void**) &gridIndex, cloud->pcl.count * sizeof(uint) ) );
    checkCudaErrors( hipMemset( gridHash, 0, cloud->pcl.count * sizeof(uint) ) );
    checkCudaErrors( hipMemset( gridIndex, 0, cloud->pcl.count * sizeof(uint) ) );

    calcHashD<<< numBlocks, numThreads >>>( gridHash,
                                            gridIndex,
                                            d_pos );

    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");

    thrust::sort_by_key(thrust::device_ptr<uint>(gridHash),
                            thrust::device_ptr<uint>(gridHash + cloud->pcl.count),
                            thrust::device_ptr<uint>(gridIndex));

    checkCudaErrors( hipMemset( cellStart, 0xffffffff, cloud->world.count * sizeof(uint) ) );
    checkCudaErrors( hipMemset( cellEnd, 0xffffffff, cloud->world.count * sizeof(uint) ) );

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>( cellStart,
                                                                        cellEnd,
                                                                        gridHash,
                                                                        gridIndex );

    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");


    checkCudaErrors( hipMemset( d_red, 0, cloud->world.count ) );
    checkCudaErrors( hipMemset( d_green, 0, cloud->world.count ) );
    checkCudaErrors( hipMemset( d_blue, 0, cloud->world.count ) );
    cuda_create_color_maps<<< numBlocks, numThreads >>> ( d_pos,
                                                          d_color,
                                                          gridIndex,
                                                          cellStart,
                                                          cellEnd,
                                                          d_red,
                                                          d_green,
                                                          d_blue );
    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");

    checkCudaErrors( hipMemcpy3D( &redParams ) );
    checkCudaErrors( hipMemcpy3D( &greenParams ) );
    checkCudaErrors( hipMemcpy3D( &blueParams ) );

    checkCudaErrors( hipFree( d_pos ) );
    checkCudaErrors( hipFree( d_color ) );
    checkCudaErrors( hipFree( gridHash ) );
    checkCudaErrors( hipFree( gridIndex ) );

    delete [] h_pos;
    delete [] h_color;
}


extern "C"
void freeCudaBuffers()
{
    checkCudaErrors( hipFree( cellEnd ) );
    checkCudaErrors( hipFree( cellStart ) );

    checkCudaErrors( hipUnbindTexture(texRed) );
    checkCudaErrors( hipFreeArray(d_redArray) );

    checkCudaErrors( hipUnbindTexture(texGreen) );
    checkCudaErrors( hipFreeArray(d_greenArray) );

    checkCudaErrors( hipUnbindTexture(texBlue) );
    checkCudaErrors( hipFreeArray(d_blueArray) );

    checkCudaErrors( hipFree( d_red ) );
    checkCudaErrors( hipFree( d_green ) );
    checkCudaErrors( hipFree( d_blue ) );

    checkCudaErrors( hipFree( d_volume ) );
}


extern "C"
void render_kernel( dim3 gridSize, dim3 blockSize,
                    unsigned char *buffer,
                    uint imageW, uint imageH,
                    float dens, float bright, float offset, float scale,
                    float *fps )
{
    float cudatime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    checkCudaErrors( hipMemset( d_volume, 0, imageW * imageH * 3 ) );
    d_render<<<gridSize,blockSize>>>( d_volume,
                                      imageW, imageH,
                                      dens, bright, offset, scale );
    hipDeviceSynchronize();
    checkCudaErrors( hipMemcpy( buffer, d_volume, imageW * imageH * 3, hipMemcpyDeviceToHost ) );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &cudatime, start, stop );
    *fps = cudatime;
}


extern "C"
void copyInvViewMatrix( float *invViewMatrix, size_t sizeofMatrix )
{
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( c_invViewMatrix), invViewMatrix, sizeofMatrix, 0, hipMemcpyHostToDevice ) );
}




